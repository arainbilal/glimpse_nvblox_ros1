#include "hip/hip_runtime.h"
#include <nvblox/core/bounding_boxes.h>
#include <nvblox/core/unified_vector.h>
#include <nvblox/gpu_hash/cuda/gpu_hash_interface.cuh>
#include <nvblox/gpu_hash/cuda/gpu_indexing.cuh>

#include <thrust/functional.h>
#include <thrust/transform.h>

#include "nvblox_ros/conversions.hpp"

namespace nvblox {

template <typename VoxelType>
__device__ bool getVoxelIntensity(const VoxelType& voxel, float voxel_size,
                                  float* intensity);

template <>
__device__ bool getVoxelIntensity(const EsdfVoxel& voxel, float voxel_size,
                                  float* intensity) {
  *intensity = voxel_size * sqrtf(voxel.squared_distance_vox);
  if (voxel.is_inside) {
    *intensity = -*intensity;
  }
  return voxel.observed;
}

template <>
__device__ bool getVoxelIntensity(const TsdfVoxel& voxel, float voxel_size,
                                  float* intensity) {
  constexpr float kMinWeight = 0.1f;
  *intensity = voxel.distance;
  return voxel.weight > kMinWeight;
}

// Inputs: GPU hash for the E/TSDF.
//         AABB.
//         Voxel Size (just needed for ESDF).
// Outputs: vector of pcl::PointXYZIs.
//          max index (updated atomically).
template <typename VoxelType>
__global__ void copyPointcloudToPCL(
    Index3DDeviceHashMapType<VoxelBlock<VoxelType>> block_hash,
    Index3D* block_indices, size_t num_indices, int max_output_indices,
    AxisAlignedBoundingBox aabb, float block_size, PclPoint* pointcloud,
    int* max_index) {
  const float voxel_size = block_size / VoxelBlock<VoxelType>::kVoxelsPerSide;

  // Get the relevant block.
  __shared__ VoxelBlock<VoxelType>* block_ptr;
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    block_ptr = nullptr;
    auto it = block_hash.find(block_indices[blockIdx.x]);
    if (it != block_hash.end()) {
      block_ptr = it->second;
    } else {
      return;
    }
  }

  __syncthreads();

  if (block_ptr == nullptr) {
    return;
  }

  // For every voxel, check if it's in the AABB.
  Index3D voxel_index(threadIdx.x, threadIdx.y, threadIdx.z);

  // Get the voxel position:
  Vector3f voxel_position = getPositionFromBlockIndexAndVoxelIndex(
      block_size, block_indices[blockIdx.x], voxel_index);

  if (!aabb.contains(voxel_position)) {
    return;
  }

  // Check if this voxel sucks or not.
  const VoxelType& voxel =
      block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  float intensity = 0.0f;
  if (!getVoxelIntensity<VoxelType>(voxel, voxel_size, &intensity)) {
    return;
  }

  // Otherwise shove it in the output.
  int next_index = atomicAdd(max_index, 1);
  if (next_index >= max_output_indices) {
    printf("Overrunning the space. This shouldn't happen.\n");
    return;
  }
  PclPoint& point = pointcloud[next_index];
  point.x = voxel_position.x();
  point.y = voxel_position.y();
  point.z = voxel_position.z();
  point.intensity = intensity;
}

template <typename VoxelType>
void RosConverter::convertLayerInAABBToPCLCuda(
    const VoxelBlockLayer<VoxelType>& layer, const AxisAlignedBoundingBox& aabb,
    sensor_msgs::PointCloud2* pointcloud) {
  constexpr int kVoxelsPerSide = VoxelBlock<TsdfVoxel>::kVoxelsPerSide;
  constexpr int kVoxelsPerBlock =
      kVoxelsPerSide * kVoxelsPerSide * kVoxelsPerSide;
  const float voxel_size = layer.voxel_size();

  // In case the AABB is infinite, make sure we have a finite number of
  // voxels.
  AxisAlignedBoundingBox aabb_intersect = getAABBOfAllocatedBlocks(layer);
  if (!aabb.isEmpty()) {
    aabb_intersect = aabb_intersect.intersection(aabb);
  }

  // Figure out which blocks are in the AABB.
  std::vector<Index3D> block_indices =
      getAllocatedBlocksWithinAABB(layer, aabb_intersect);
  // Copy to device memory.
  block_indices_device_ = block_indices;

  if (block_indices.empty()) {
    return;
  }
  size_t num_voxels = block_indices.size() * kVoxelsPerBlock;

  // Allocate a GPU pointcloud.
  pointcloud_device_.reserve(num_voxels);

  // Get the hash.
  GPULayerView<VoxelBlock<VoxelType>> gpu_layer_view = layer.getGpuLayerView();

  // Create an output size variable.
  if (!max_index_device_) {
    max_index_device_ = make_unified<int>(MemoryType::kDevice);
  }
  max_index_device_.setZero();

  // Call the kernel.
  int dim_block = block_indices.size();
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);

  copyPointcloudToPCL<VoxelType><<<dim_block, dim_threads, 0, cuda_stream_>>>(
      gpu_layer_view.getHash().impl_, block_indices_device_.data(),
      block_indices.size(), num_voxels, aabb_intersect, layer.block_size(),
      pointcloud_device_.data(), max_index_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());

  // Copy the pointcloud out.
  max_index_host_ = max_index_device_.clone(MemoryType::kHost);

  size_t output_size = sizeof(PclPoint) * *max_index_host_;
  pointcloud->data.resize(output_size);
  // Copy over all the points.
  hipMemcpy(pointcloud->data.data(), pointcloud_device_.data(), output_size,
             hipMemcpyDeviceToHost);

  // Fill the other fields in the pointcloud message.
  pointcloud->height = 1;
  pointcloud->width = *max_index_host_;
  pointcloud->point_step = sizeof(PclPoint);
  pointcloud->row_step = output_size;

  // Populate the fields.
  sensor_msgs::PointField point_field;
  point_field.name = "x";
  point_field.datatype = sensor_msgs::PointField::FLOAT32;
  point_field.offset = 0;
  point_field.count = 1;

  pointcloud->fields.push_back(point_field);
  point_field.name = "y";
  point_field.offset += sizeof(float);
  pointcloud->fields.push_back(point_field);
  point_field.name = "z";
  point_field.offset += sizeof(float);
  pointcloud->fields.push_back(point_field);
  point_field.name = "intensity";
  point_field.offset += sizeof(float);
  pointcloud->fields.push_back(point_field);
}

// Template specializations.
template void RosConverter::convertLayerInAABBToPCLCuda<TsdfVoxel>(
    const VoxelBlockLayer<TsdfVoxel>& layer, const AxisAlignedBoundingBox& aabb,
    sensor_msgs::PointCloud2* pointcloud);

template void RosConverter::convertLayerInAABBToPCLCuda<EsdfVoxel>(
    const VoxelBlockLayer<EsdfVoxel>& layer, const AxisAlignedBoundingBox& aabb,
    sensor_msgs::PointCloud2* pointcloud);

void RosConverter::meshBlockMessageFromMeshBlock(
    const MeshBlock& mesh_block, nvblox_msgs::MeshBlock* mesh_block_msg) {
  CHECK_NOTNULL(mesh_block_msg);

  size_t num_vertices = mesh_block.vertices.size();

  mesh_block_msg->vertices.resize(num_vertices);
  mesh_block_msg->normals.resize(num_vertices);
  mesh_block_msg->colors.resize(mesh_block.colors.size());
  mesh_block_msg->triangles.resize(mesh_block.triangles.size());

  std::vector<Vector3f> vertices = mesh_block.getVertexVectorOnCPU();
  std::vector<Vector3f> normals = mesh_block.getNormalVectorOnCPU();
  std::vector<Color> colors = mesh_block.getColorVectorOnCPU();

  // Copy over vertices and normals.
  for (size_t i = 0; i < num_vertices; i++) {
    mesh_block_msg->vertices[i] = pointMessageFromVector(vertices[i]);
    mesh_block_msg->normals[i] = pointMessageFromVector(normals[i]);
  }

  // Copy over colors if available.
  for (size_t i = 0; i < mesh_block.colors.size(); i++) {
    mesh_block_msg->colors[i] = colorMessageFromColor(colors[i]);
  }

  // Copying over triangles is thankfully easy.
  mesh_block_msg->triangles = mesh_block.getTriangleVectorOnCPU();
}

struct DivideBy1000 : public thrust::unary_function<uint16_t, float> {
  __host__ __device__ float operator()(const uint16_t& in) {
    return static_cast<float>(in) / 1000.0f;
  }
};

// Convert image to depth frame object
bool RosConverter::depthImageFromImageMessage(
    const sensor_msgs::ImageConstPtr& image_msg,
    DepthImage* depth_image) {
  CHECK_NOTNULL(depth_image);
  // If the image is a float, we can just copy it over directly.
  // If the image is int16, we need to divide by 1000 to get the correct
  // format for us.
  
  /*
  std::cout << "Processing depth image" << std::endl;
  std::cout << "Processing depth image data is " << &image_msg->data[0] << std::endl;
  std::cout << "Processing depth image encoding is " << image_msg->encoding << std::endl;
  std::cout << "Processing depth image step is " << image_msg->step << std::endl;
  std::cout << "Processing depth image width is " << image_msg->width << std::endl;
  std::cout << "Processing depth image height is " << image_msg->height << std::endl;
  std::cout << "SizeOf Uint16 is " << sizeof(uint16_t) << std::endl;
  std::cout << "SizeOf Uint8 is " << sizeof(uint8_t) << std::endl;
*/

  // First check if we actually have a valid image here.
  if (image_msg->encoding != "32FC1" && image_msg->encoding != "16UC1") {
    return false;
  }

  // Fill it in. How this is done depends on what the image encoding is.
  if (image_msg->encoding == "32FC1") {
    // Float to float, so this should be a straight-up copy. :)
    depth_image->populateFromBuffer(
        image_msg->height, image_msg->width,
        reinterpret_cast<const float*>(&image_msg->data[0]));
  } else if (image_msg->encoding == "16UC1") {
    
    // Then we have to just go byte-by-byte and convert this. This is a massive
    // pain and slow. We need to find a better way to do this; on GPU or
    // through openCV.
    const uint16_t* char_depth_buffer =
        reinterpret_cast<const uint16_t*>(&image_msg->data[0]);
    
    const int intWidth = image_msg->width;
    const int intHeight = image_msg->height;

    const int numel = intWidth * intHeight;
    
    bool kUseCuda = false;
    if (kUseCuda) {
      
      // Make sure there's enough output space.
      if (depth_image->numel() < numel) {
        *depth_image = DepthImage(image_msg->height, image_msg->width,
                                  MemoryType::kDevice);
      }

      // Now just thrust it.
      thrust::transform(char_depth_buffer, char_depth_buffer + numel, depth_image->dataPtr(), DivideBy1000());
    } else {
      std::vector<float> float_depth_buffer(numel);
      
      for (int i = 0; i < numel; i++) {
        float_depth_buffer[i] =
            static_cast<float>(char_depth_buffer[i]) / 1000.0f;
            if(!std::isfinite(float_depth_buffer[i])){
            }
      }
      depth_image->populateFromBuffer(intHeight, intWidth,
                                      float_depth_buffer.data(),
                                      MemoryType::kDevice);
    }
  }

  return true;
}

__global__ void populateSliceFromLayerKernel(
    Index3DDeviceHashMapType<EsdfBlock> block_hash, AxisAlignedBoundingBox aabb,
    float block_size, float* image, int rows, int cols, float z_slice_height,
    float resolution, float unobserved_value) {
  const float voxel_size = block_size / EsdfBlock::kVoxelsPerSide;
  const int pixel_col = blockIdx.x * blockDim.x + threadIdx.x;
  const int pixel_row = blockIdx.y * blockDim.y + threadIdx.y;

  if (pixel_col >= cols || pixel_row >= rows) {
    return;
  }

  // Figure out where this pixel should map to.
  Vector3f voxel_position(aabb.min().x() + resolution * pixel_col,
                          aabb.min().y() + resolution * pixel_row,
                          z_slice_height);

  Index3D block_index, voxel_index;

  getBlockAndVoxelIndexFromPositionInLayer(block_size, voxel_position,
                                           &block_index, &voxel_index);

  // Get the relevant block.
  EsdfBlock* block_ptr = nullptr;
  auto it = block_hash.find(block_index);
  if (it != block_hash.end()) {
    block_ptr = it->second;
  } else {
    image::access(pixel_row, pixel_col, cols, image) = unobserved_value;
    return;
  }

  // Get the relevant pixel.
  const EsdfVoxel* voxel =
      &block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  float distance = unobserved_value;
  if (voxel->observed) {
    distance = voxel_size * std::sqrt(voxel->squared_distance_vox);
    if (voxel->is_inside) {
      distance = -distance;
    }
  }
  image::access(pixel_row, pixel_col, cols, image) = distance;
}

void RosConverter::populateSliceFromLayer(const EsdfLayer& layer,
                                          const AxisAlignedBoundingBox& aabb,
                                          float z_slice_height,
                                          float resolution,
                                          float unobserved_value,
                                          Image<float>* image) {
  if (image->numel() <= 0) {
    return;
  }
  const float voxel_size = layer.voxel_size();

  // Create a GPU hash of the ESDF.
  GPULayerView<EsdfBlock> gpu_layer_view = layer.getGpuLayerView();

  // Pass in the GPU hash and AABB and let the kernel figure it out.
  constexpr int kThreadDim = 16;
  const int rounded_rows = static_cast<int>(
      std::ceil(image->rows() / static_cast<float>(kThreadDim)));
  const int rounded_cols = static_cast<int>(
      std::ceil(image->cols() / static_cast<float>(kThreadDim)));
  dim3 block_dim(rounded_cols, rounded_rows);
  dim3 thread_dim(kThreadDim, kThreadDim);

  populateSliceFromLayerKernel<<<block_dim, thread_dim, 0, cuda_stream_>>>(
      gpu_layer_view.getHash().impl_, aabb, layer.block_size(),
      image->dataPtr(), image->rows(), image->cols(), z_slice_height,
      resolution, unobserved_value);
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());
}

__global__ void populateCloudFromImageKernel(const float* image,
                                             int rows,
                                             int cols,
                                             const Camera camera,
                                             int raycast_subsampling_factor,
                                             PclPoint* pointcloud,
                                             int* max_index)
{
    const int ray_idx_row = blockIdx.x * blockDim.x + threadIdx.x;
    const int ray_idx_col = blockIdx.y * blockDim.y + threadIdx.y;
    int pixel_row = ray_idx_row * raycast_subsampling_factor;
    int pixel_col = ray_idx_col * raycast_subsampling_factor;

    // Hooray we do nothing.
    if (pixel_row >= (rows + raycast_subsampling_factor - 1) ||
        pixel_col >= (cols + raycast_subsampling_factor - 1))
    {
      return;
    }
    else
    {
      // Move remaining overhanging pixels back to the borders.
      if (pixel_row >= rows)
      {
        pixel_row = rows - 1;
      }
      if (pixel_col >= cols)
      {
        pixel_col = cols - 1;
      }
    }

    float depth = image::access<float>(pixel_row, pixel_col, cols, image);
    if ((depth <= 0.0f) || std::isnan(depth))
    {
      return;
    }
    else
    {
        ///printf("(%d,%d) %f\n",pixel_row, pixel_col, depth);
        /// Copy it to the PCL
        float center_x = camera.cu();//model.cx();
        float center_y = camera.cv();//model.cy();
        float constant_x = 1.0 / camera.fu();//model.fx();
        float constant_y = 1.0 / camera.fv();//model.fy();
        //float bad_point = std::numeric_limits<float>::quiet_NaN();

        int next_index = atomicAdd(max_index, 1);
        printf("%d\n", next_index);
#if(0)
        PclPoint& point = pointcloud[next_index];
        point.x = (pixel_row - center_x) * depth * constant_x;
        point.y = (pixel_col - center_y) * depth * constant_y;
        point.z = depth;
        point.intensity = 0.0;
#endif
    }

}

#if(0)
void RosConverter::pointcloudFromDepth(const DepthImage& depth_frame, const Camera& camera, sensor_msgs::PointCloud2* pointcloud)
{
    /// Reference: nvblox_ros/src/lib/conversions/pointcloud_conversion.cu
    const int num_points = depth_frame.rows() * depth_frame.cols();
    std::cout<< "Number of points: " << num_points << std::endl;

    /// Copy the depth points into pinned host memory
    depth_points_host_.clear();
    float center_x = camera.cu();//model.cx();
    float center_y = camera.cv();//model.cy();
    float constant_x = 1.0 / camera.fu();//model.fx();
    float constant_y = 1.0 / camera.fv();//model.fy();

}
#endif

#if(1)
void RosConverter::pointcloudFromDepth(const DepthImage& depth_frame, const Camera& camera, sensor_msgs::PointCloud2* pointcloud)
{
    /// Reference: See frustum.cu in nvblox
    unsigned int raycast_subsampling_factor_ = 1;
    const int num_subsampled_rows = std::ceil(static_cast<float>(depth_frame.rows() + 1)/static_cast<float>(raycast_subsampling_factor_));
    const int num_subsampled_cols = std::ceil(static_cast<float>(depth_frame.cols() + 1) /static_cast<float>(raycast_subsampling_factor_));
    /// We'll do warps of 32x32 pixels in the image. This is 1024 threads which is in the recommended 512-1024 range.
    constexpr int kThreadDim = 16;
    const int rounded_rows = static_cast<int>(std::ceil(num_subsampled_rows / static_cast<float>(kThreadDim)));
    const int rounded_cols = static_cast<int>(std::ceil(num_subsampled_cols / static_cast<float>(kThreadDim)));

    dim3 block_dim(rounded_rows, rounded_cols);
    dim3 thread_dim(kThreadDim, kThreadDim);
    std::cout << "Processing point cloud from depth image" << std::endl;
    std::cout << "Rounded rows: " << rounded_rows << std::endl;
    std::cout << "Rounded rows: " << rounded_cols << std::endl;
    std::cout << "Threds dimension: " << kThreadDim << std::endl;


    /// Create an output size variable.
    if (!max_index_device_)
    {
        max_index_device_ = make_unified<int>(MemoryType::kDevice);
    }
    max_index_device_.setZero();

    /// Allocate a GPU pointcloud.
    size_t num_points = depth_frame.rows()*depth_frame.cols();

    pointcloud_device_.reserve(num_points);

    /// !!! Check block_dim, thread_dim and pointcloud_device_.reserve!!!!
    populateCloudFromImageKernel<<<block_dim, thread_dim, 0, cuda_stream_>>>(
      depth_frame.dataConstPtr(),depth_frame.rows(),depth_frame.cols(),camera,
      raycast_subsampling_factor_,pointcloud_device_.data(), max_index_device_.get());

    checkCudaErrors(hipStreamSynchronize(cuda_stream_));
    checkCudaErrors(hipPeekAtLastError());

#if(0)
      /// Copy the pointcloud out
      max_index_host_ = max_index_device_.clone(MemoryType::kHost);

      size_t output_size = sizeof(PclPoint) * *max_index_host_;
      pointcloud->data.resize(output_size);

      // Copy over all the points.
      hipMemcpy(pointcloud->data.data(), pointcloud_device_.data(), output_size, hipMemcpyDeviceToHost);

      std::cout << "Output size: " << output_size << std::endl;
      // Fill the other fields in the pointcloud message.
      pointcloud->height = 1;
      pointcloud->width = *max_index_host_;
      pointcloud->point_step = sizeof(PclPoint);
      pointcloud->row_step = output_size;

      // Populate the fields.
      sensor_msgs::PointField point_field;
      point_field.name = "x";
      point_field.datatype = sensor_msgs::PointField::FLOAT32;
      point_field.offset = 0;
      point_field.count = 1;

      pointcloud->fields.push_back(point_field);
      point_field.name = "y";
      point_field.offset += sizeof(float);
      pointcloud->fields.push_back(point_field);
      point_field.name = "z";
      point_field.offset += sizeof(float);
      pointcloud->fields.push_back(point_field);
      point_field.name = "intensity";
      point_field.offset += sizeof(float);
      pointcloud->fields.push_back(point_field);
#endif
}
#endif

}  // namespace nvblox
